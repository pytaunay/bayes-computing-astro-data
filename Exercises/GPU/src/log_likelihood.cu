#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>

#define M_PI 3.14159265358979323846

// For CUBLAS
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

// Calculates (X-mu)^T * ISIGMA * (X-mu) for nobs random variables of size sizex
// Stores the result in res
// Intemediary vectors LV, tmp, ones are allocated before the function is called
//
// The routine uses the GPU analog of BLAS GEMM, GEMV, AXPY, and DDOT for high-perf linear algebra operations
// The library to do BLAS on the GPU is ... CUBLAS. Note that it is expecting column-major matrices, which changes the different calls to
// GEMM, GEMV, etc.
void log_likelihood(double *X, double *isigma, double *mu, double det_sig, const int nobs,const int sizex, double *res, double *LV, double *tmp, double *ones, hipblasHandle_t *handle) {


	hipblasStatus_t stat;
	double ZERO = 0.0;
	double ONE = 1.0;
	double NONE = -1.0;

	// Thrust is a library for developer productivity, analog of the C++ standard library
	// Here we define pointers that reside in the device memory: device_ptr
	thrust::device_ptr<double> d_ones = thrust::device_pointer_cast(ones);
	thrust::device_ptr<double> d_mu = thrust::device_pointer_cast(mu);
	thrust::device_ptr<double> d_LV = thrust::device_pointer_cast(LV);
	thrust::device_ptr<double> d_tmp = thrust::device_pointer_cast(tmp);

	// We can them conveniently use them through readily available routines such as fill, copy, transform, etc
	// Fill "ones" with 1
	thrust::fill(d_ones, d_ones+nobs, ONE);

	if( nobs > 1) {
	//	cblas_dgemm(CblasRowMajor,CblasTrans,CblasNoTrans,nobs,sizex,1,1.0,ones,nobs,mu,sizex,0.0,LV,sizex); 
		stat = hipblasDgemm(*handle,HIPBLAS_OP_N,HIPBLAS_OP_T,sizex,nobs,1,&ONE,mu,sizex,ones,nobs,&ZERO,LV,sizex); 
		if( stat != HIPBLAS_STATUS_SUCCESS ) {
			printf("CUBLAS DGEMM failed\n");
			exit(0);
		}	
	} else {	
		thrust::copy(d_mu,d_mu+sizex,d_LV);
	}

	// Calculate LV = (X-mu)
	// We store mu-X in LV bc. of AXPY: don't want to write over X 
	// cblas_daxpy(nobs*sizex,-1.0,X,1,LV,1);
	stat = hipblasDaxpy(*handle,nobs*sizex,&NONE,X,1,LV,1);
	if( stat != HIPBLAS_STATUS_SUCCESS ) {
		printf("CUBLAS DAXPY failed\n");
		exit(0);
	}	
	
	if( nobs > 1 ) {
		// Calculate LV*SIGMA: [NOBS x SIZEX] x [SIZEX x SIZEX]
		//cblas_dgemm(CblasRowMajor,CblasNoTrans,CblasNoTrans,nobs,sizex,sizex,1.0,LV,sizex,isigma,sizex,0.0,tmp,sizex); 
		stat = hipblasDgemm(*handle,HIPBLAS_OP_T,HIPBLAS_OP_N,sizex,nobs,sizex,&ONE,isigma,sizex,LV,sizex,&ZERO,tmp,sizex);
		if( stat != HIPBLAS_STATUS_SUCCESS ) {
			printf("CUBLAS DGEMM failed\n");
			exit(0);
		}	
	} else {
		// If there is only one observation, a gemv is sufficient: 
		// [1 x SIZEX] x [SIZEX x SIZEX]
		// cblas_dgemv(CblasRowMajor,CblasTrans,sizex,sizex,1.0,isigma,sizex,LV,1,0.0,tmp,1);
		stat = hipblasDgemv(*handle,HIPBLAS_OP_T,sizex,sizex,&ONE,isigma,sizex,LV,1,&ZERO,tmp,1);
		if( stat != HIPBLAS_STATUS_SUCCESS ) {
			printf("CUBLAS DGEMV failed\n");
			exit(0);
		}	
	}	

	// Calculate [LV*SIGMA]*RV = tmp*RV: dot product
	// We use LV as RV, with increments between elements equal to 1, making it a column vector
	double sum = 0.0;
	// sum = cblas_ddot(sizex*nobs,tmp,1,LV,1);	
	stat = hipblasDdot(*handle,sizex*nobs,tmp,1,LV,1,&sum);
	if( stat != HIPBLAS_STATUS_SUCCESS ) {
		printf("CUBLAS DDOT failed: %d\n",stat);
		exit(0);
	}	

	// Finish 
	sum += (double)nobs*(double)sizex*log(2.0*M_PI) + (double)nobs*log(fabs(det_sig));
	sum *= (-0.5);

	*res = sum;
}

