#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "read_data.h"
#include "log_likelihood.h"
#include "tools.h"

int main(int argc, char *argv[]) {

	int nobs, sizex, nsample = 0;
	char *location = NULL;
	int ret = 0;

	//////////////////////////////////////
	/////////////// PARSERS //////////////
	//////////////////////////////////////
	// Parse the command line
	ret = parse_command_line(argc,argv,&nobs,&sizex,&nsample,&location);
	if( ret != PARSER_SUCCESS ) {
		printf("Parsing failed ! Exiting...\n");
		return EXIT_FAILURE;
	}

	// Parse the data on master
	double *X = (double*)malloc(nobs*sizex*sizeof(double));	
	double *isigma = (double*)malloc(sizex*sizex*sizeof(double));
	double *mu = (double*)malloc(sizex*sizeof(double));
	double det_sigma = 0.0; 

	ret = read_data(X, isigma, &det_sigma, mu, &nobs, &sizex, location);
	if( ret != PARSER_SUCCESS ) {
		printf("Parsing failed ! Exiting...\n");
		return EXIT_FAILURE;
	}	

	////////////////////////////////////////
	/////////////// Variables //////////////
	////////////////////////////////////////
	// Timing variables
	double tic, toc, tot_time = 0.0; 

	// Result
	double res = 0.0;
	

	// Allocate GPU memory
	double *d_LV, *d_tmp, *d_ones;
	hipMalloc((void**)&d_LV,nobs*sizex*sizeof(double));
	hipMalloc((void**)&d_tmp,nobs*sizex*sizeof(double));
	hipMalloc((void**)&d_ones,nobs*sizeof(double));

	double *d_X, *d_isigma, *d_mu;
	hipMalloc((void**)&d_X,nobs*sizex*sizeof(double));
	hipMalloc((void**)&d_isigma,sizex*sizex*sizeof(double));
	hipMalloc((void**)&d_mu,sizex*sizeof(double));

	// Copy the read data onto the GPU
	hipMemcpy(d_X,X,nobs*sizex*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_isigma,isigma,sizex*sizex*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_mu,mu,sizex*sizeof(double),hipMemcpyHostToDevice);

	// Create a handle for cublas, the GPU linear algebra library
	hipblasHandle_t handle;
	hipblasStatus_t stat;
	stat = hipblasCreate(&handle);
	hipblasSetPointerMode(handle,HIPBLAS_POINTER_MODE_HOST);
	if( stat != HIPBLAS_STATUS_SUCCESS ) {
		printf("CUBLAS Initialization failed\n");
		return EXIT_FAILURE;
	}	


	////////////////////////////////////////
	/////////////// Algorithm //////////////
	////////////////////////////////////////
	//// Start time sampling
	for(int k = 0; k < nsample; k++) {
		tic = omp_get_wtime();
		res  = 0.0;

		// Main driver
		log_likelihood(d_X,d_isigma,d_mu,det_sigma,nobs,sizex,&res,d_LV,d_tmp,d_ones,&handle);

		toc = omp_get_wtime();
		tot_time += toc-tic;
	}	

	printf("Result: %f\n",res);
	printf("Total time: %f\n",tot_time/(double)nsample);

	////////////////////////////////////////
	/////////////// Clean up ///////////////
	////////////////////////////////////////
	hipFree(d_mu);
	hipFree(d_X);
	hipFree(d_isigma);
	hipFree(d_LV);
	hipFree(d_tmp);
	hipFree(d_ones);

	free(X);
	free(isigma);
	free(mu);
	free(location);

	return EXIT_SUCCESS;
}
